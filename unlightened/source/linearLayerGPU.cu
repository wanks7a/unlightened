#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <LinearLayerGPU.h>
#include <GpuUtils.h>
#include <algorithm>

#define trPerBlock 256

template <typename T, bool BIAS_NOT_INCLUDED>
__global__ void k_linearLayerForwardPass(T* output,const T* weights, const T* input, size_t inputSize, size_t outputSize)
{
    const unsigned int batch_offset_output = BIAS_NOT_INCLUDED ? blockIdx.y * (outputSize + 1) : blockIdx.y * outputSize;
    const unsigned int batch_offset_input = blockIdx.y * inputSize;
    const auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < outputSize)
    {
        float result = 0.0f;
        for (int j = 0; j < inputSize; j++)
        {
            //result = __fmaf_rn(input[j], weights[i * inputSize + j], result); // very fast multiply add = a*b + c
            result += input[batch_offset_input + j] * weights[i * inputSize + j];
        }
        output[batch_offset_output + i] = result;
    }
}

void linearLayerForwardPassGPU(float* output,const float* weights, const float* input, const shape& input_shape, const shape& output_shape, bool bias_subtracted)
{
    auto threadsPerBlock = static_cast<unsigned int>(std::min(output_shape.width, static_cast<size_t>(trPerBlock)));
    auto num_of_blocks = utils::getBlockSize(threadsPerBlock, output_shape.width);
    dim3 blocks(num_of_blocks, output_shape.batches);
    if(bias_subtracted)
        k_linearLayerForwardPass<float, true> << <blocks, threadsPerBlock >> > (output, weights, input, input_shape.width, output_shape.width);
    else
        k_linearLayerForwardPass<float, false> << <blocks, threadsPerBlock >> > (output, weights, input, input_shape.width, output_shape.width);
    utils::waitAndCheckForErrors();
}

template <typename T>
__global__ void k_calcDerivativeWRtoInput(T* derivativeWRtoInput, size_t inputSize, const T* derivateWRtoOutput, size_t outputSize, const T* weights)
{
    auto inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (inputIndex < inputSize)
    {
        derivativeWRtoInput[inputIndex] = 0.0f;
        for (size_t i = 0; i < outputSize; i++)
        {
            derivativeWRtoInput[inputIndex] += derivateWRtoOutput[i] * weights[i * inputSize + inputIndex];
        }
    }
}

void calcDerivativeWRtoInput(float* derivativeWRtoInput, size_t inputSize, const float* derivateWRtoOutput, size_t outputSize, const float* weights)
{
    auto threadsPerBlock = static_cast<unsigned int>(std::min(inputSize, static_cast<size_t>(trPerBlock)));
    auto blocks = utils::getBlockSize(threadsPerBlock, inputSize);
    k_calcDerivativeWRtoInput << <blocks, threadsPerBlock >> > (derivativeWRtoInput, inputSize, derivateWRtoOutput, outputSize, weights);
    utils::waitAndCheckForErrors();
}

template <typename T>
__global__ void k_updateWeightsAndBias(T* weights, const T* derivativeWRtoOutput,const T* input, size_t inputSize, size_t outputSize)
{
    float learning_rate = 0.1f;
    size_t neuronIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if( neuronIndex < outputSize )
    {
        for (size_t i = 0; i < inputSize; i++)
        {
            weights[neuronIndex * inputSize + i] = weights[neuronIndex * inputSize + i] - learning_rate * input[i] * derivativeWRtoOutput[neuronIndex];
        }
    }
}

void updateWeightsAndBias(float* weights, const float* derivativeWRtoOutput, const float* input, size_t inputSize, size_t outputSize)
{
    auto threadsPerBlock = static_cast<unsigned int>(std::min(outputSize, static_cast<size_t>(trPerBlock)));
    auto blocks = utils::getBlockSize(threadsPerBlock, outputSize);
    k_updateWeightsAndBias << <blocks, threadsPerBlock >> > (weights, derivativeWRtoOutput, input, inputSize, outputSize);
    utils::waitAndCheckForErrors();
}