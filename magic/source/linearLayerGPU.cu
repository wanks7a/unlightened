#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <LinearLayerGPU.h>
#include <GpuUtils.h>
#include <algorithm>

#define trPerBlock 256

template <typename T>
__global__ void k_linearLayerForwardPass(T* output, T* weights, const T* input, size_t inputSize, size_t outputSize)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < outputSize)
    {
        float result = 0.0f;
        for (int j = 0; j < inputSize; j++)
        {
            __fmaf_rn(input[j], weights[i * inputSize + j], result); // very fast multiply add = a*b + c
            //result += input[j] * weights[i * inputSize + j];
        }
        output[i] = result;
    }
}

void linearLayerForwardPassGPU(float* output, float* weights, const float* input, size_t inputSize, size_t outputSize)
{
    auto threadsPerBlock = static_cast<unsigned int>(std::min(outputSize, static_cast<size_t>(trPerBlock)));
    auto blocks = utils::getBlockSize(threadsPerBlock, outputSize);
    k_linearLayerForwardPass << <blocks, threadsPerBlock >> > (output, weights, input, inputSize, outputSize);
    utils::waitAndCheckForErrors();
}

template <typename T>
__global__ void k_calcDerivativeWRtoInput(T* derivativeWRtoInput, size_t inputSize, const T* derivateWRtoOutput, size_t outputSize, const T* weights)
{
    auto inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (inputIndex < inputSize)
    {
        derivativeWRtoInput[inputIndex] = 0.0f;
        for (size_t i = 0; i < outputSize; i++)
        {
            derivativeWRtoInput[inputIndex] += derivateWRtoOutput[i] * weights[i * inputSize + inputIndex];
        }
    }
}

void calcDerivativeWRtoInput(float* derivativeWRtoInput, size_t inputSize, const float* derivateWRtoOutput, size_t outputSize, const float* weights)
{
    auto threadsPerBlock = static_cast<unsigned int>(std::min(inputSize, static_cast<size_t>(trPerBlock)));
    auto blocks = utils::getBlockSize(threadsPerBlock, inputSize);
    k_calcDerivativeWRtoInput << <blocks, threadsPerBlock >> > (derivativeWRtoInput, inputSize, derivateWRtoOutput, outputSize, weights);
    utils::waitAndCheckForErrors();
}

template <typename T>
__global__ void k_updateWeightsAndBias(T* weights, const T* derivativeWRtoOutput,const T* input, size_t inputSize, size_t outputSize)
{
    float learning_rate = 0.1f;
    size_t neuronIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if( neuronIndex < outputSize )
    {
        for (size_t i = 0; i < inputSize; i++)
        {
            weights[neuronIndex * inputSize + i] = weights[neuronIndex * inputSize + i] - learning_rate * input[i] * derivativeWRtoOutput[neuronIndex];
        }
    }
}

void updateWeightsAndBias(float* weights, const float* derivativeWRtoOutput, const float* input, size_t inputSize, size_t outputSize)
{
    auto threadsPerBlock = static_cast<unsigned int>(std::min(outputSize, static_cast<size_t>(trPerBlock)));
    auto blocks = utils::getBlockSize(threadsPerBlock, outputSize);
    k_updateWeightsAndBias << <blocks, threadsPerBlock >> > (weights, derivativeWRtoOutput, input, inputSize, outputSize);
    utils::waitAndCheckForErrors();
}