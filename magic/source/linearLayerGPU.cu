#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <LinearLayerGPU.h>
#include <GpuUtils.h>
#include <algorithm>

static void waitAndCheckForErrors()
{
    // Check for any errors launching the kernel
    auto cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
}

template <typename T>
__global__ void k_linearLayerForwardPass(T* output, T* weights, const T* input, size_t inputSize, size_t outputSize)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < outputSize; i += blockDim.x * gridDim.x)
    {
        output[i] = 0.0f;
        for (int j = 0; j < inputSize; j++)
        {
            output[i] += input[j] * weights[i * inputSize + j];
        }
    }
}

void linearLayerForwardPassGPU(float* output, float* weights, const float* input, size_t inputSize, size_t outputSize)
{
    auto threadsPerBlock = std::min(outputSize, static_cast<size_t>(256));
    auto blocks = utils::getBlockSize(threadsPerBlock, outputSize);
    k_linearLayerForwardPass << <blocks, threadsPerBlock >> > (output, weights, input, inputSize, outputSize);
    waitAndCheckForErrors();
}

template <typename T>
__global__ void k_calcDerivativeWRtoInput(T* derivativeWRtoInput, size_t inputSize, const T* derivateWRtoOutput, size_t outputSize, const T* weights)
{
    for (size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x; inputIndex < inputSize; inputIndex += blockDim.x * gridDim.x)
    {
        derivativeWRtoInput[inputIndex] = 0.0f;
        for (size_t i = 0; i < outputSize; i++)
        {
            derivativeWRtoInput[inputIndex] += derivateWRtoOutput[i] * weights[i * inputSize + inputIndex];
        }
    }
}

void calcDerivativeWRtoInput(float* derivativeWRtoInput, size_t inputSize, const float* derivateWRtoOutput, size_t outputSize, const float* weights)
{
    auto threadsPerBlock = std::min(inputSize, static_cast<size_t>(256));
    auto blocks = utils::getBlockSize(threadsPerBlock, inputSize);
    k_calcDerivativeWRtoInput << <blocks, threadsPerBlock >> > (derivativeWRtoInput, inputSize, derivateWRtoOutput, outputSize, weights);
    waitAndCheckForErrors();
}

template <typename T>
__global__ void k_updateWeightsAndBias(T* weights, const T* derivativeWRtoOutput,const T* input, size_t inputSize, size_t outputSize)
{
    float learning_rate = 0.1f;
    for (size_t neuronIndex = blockIdx.x * blockDim.x + threadIdx.x; neuronIndex < outputSize; neuronIndex += blockDim.x * gridDim.x)
    {
        for (size_t i = 0; i < inputSize; i++)
        {
            weights[neuronIndex * inputSize + i] = weights[neuronIndex * inputSize + i] - learning_rate * input[i] * derivativeWRtoOutput[neuronIndex];
        }
    }
}

void updateWeightsAndBias(float* weights, const float* derivativeWRtoOutput, const float* input, size_t inputSize, size_t outputSize)
{
    auto threadsPerBlock = std::min(outputSize, static_cast<size_t>(256));
    auto blocks = utils::getBlockSize(threadsPerBlock, outputSize);
    k_updateWeightsAndBias << <blocks, threadsPerBlock >> > (weights, derivativeWRtoOutput, input, inputSize, outputSize);
    waitAndCheckForErrors();
}

namespace utils
{
    size_t getBlockSize(size_t threadsPerBlock, size_t maxThreads)
    {
        return static_cast<size_t>(std::ceil(static_cast<double>(maxThreads) / threadsPerBlock));
    }

    bool GpuInit()
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        auto cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!");
            return false;
        }
        return true;
    }

    bool GpuRelase()
    {
        auto cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return false;
        }
        return true;
    }
}